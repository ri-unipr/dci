#include "hip/hip_runtime.h"

/*
* File:   kmpso.cu
* Author: Gianluigi Silvestri, Michele Amoretti, Stefano Cagnoni
*/

// SC aggiunto per usare calloc
#include <stdlib.h>

#include <iostream>
#include <cmath>
#include <numeric>
#include <algorithm>
#include <fstream>
#include <random>
#include <vector>
#include <cfloat>
#include <limits>
#include <string>
#include "application.h"
#include "kmpso_kernels.h"

using  namespace  std;

#define	D_max 500  // Max number of dimensions of the search space
#define	S_max 10000 // Max swarm size
#define K_max 3000 //Max number of clusters


// Global variables
double pi; // Useful for some test functions
int D; // Search space dimension
int S; // Swarm size
int K; // Number of seeds
int N; // number of results to keep
unsigned int rseed; // random seed
int hseed; // random seed for h. system computation
double *v; // vector for distance
double *V;
double *X;
double *P;
int *seed;
double *fx;
double *fp;
int *size;
double *M;
double *sigma;
double *bp;
double *fm;
bool *best;
vector < vector<unsigned int> > group(S_max, vector<unsigned int> (D_max));
int a;
double *d_X;
double *d_V;
double *d_P;
int *d_seed;
double *d_sigma;
double *d_bp;
hiprandState* devStates;
int TPB, NB;
double xmin, xmax; // Intervals defining the search space
int T;
vector<float> output1(S);
double x;
double c; // acceleration
double w; // constriction factor
dci::Application* app;
int fitness;
vector <double> results;
vector < vector<unsigned int> > g;
int r1, r2;
int b;

double alea( double a, double b )
{ // random number (uniform distribution) in [a b]
  double r;
  r=(double)rand(); r=r/RAND_MAX;
  return a + r * ( b - a );
}

vector<float> perf(int S, int D)
{

  // ********************************************************************* //
  // COMPUTATION SECTION - repeat as needed                                //
  // ********************************************************************* //

  // create agent list for clusters
  vector<unsigned int> cluster1(D);
  cluster1.clear();
  output1.clear();

  // allocate memory for clusters
  vector<register_t*> clusters(S);
  // allocate memory for cluster indexes
  vector<float> output(S);

  for( int s=0; s<S; s++)
  {
    group[s].clear();
    fitness++;
    for (int d=0; d<D; d++)
    {
      if (X[s*D+d]>=0)
      {
        cluster1.push_back(d);
        group[s].push_back(d);
      }
    }
    // allocate cluster bitmasks
    clusters[s] = (register_t*)malloc(app->getAgentSizeInBytes());
    // set bitmasks from agent lists
    dci::ClusterUtils::setClusterFromPosArray(clusters[s], cluster1, app->getNumberOfAgents());
    cluster1.clear();

  }

  // perform computation
  app->ComputeIndex(clusters, output);

  for (int s=0;s<S; s++)
  {
    // free memory
    free(clusters[s]);
  }

  return output;
}

void k_means()
{
  int k, d, s;
  int count=0;
  double k1, kt;
  bool change;
  bool insert;
  int seed1=-1;
  for (s=0;s<S;s++) seed[s]=-1;

  for (k=0; k<K; k++) //initialize seeds
  {
    for (d=0; d<D; d++)
    {
      M[k*D+d]= alea( xmin, xmax );
    }
    best[k]=false;
  }

  do
  {

    count++;
    change =false;
    for (k=0; k<K; k++)size[k]=0;
    for(s=0; s<S; s++) // for each particle i do
    {
      k1=0;
      insert=false; //doesn't belong to a cluster
      for (k=0; k<K; k++) // find the nearest seed mk
      {
        for (d=0; d<D; d++)
        {
          v[d] = P[s*D+d]-M[k*D+d];
        }
        kt=sqrt(inner_product(v, v+D, v, 0.0L)); // calculate distance p-m
        if((insert==false ) || kt<k1 )
        // if is the first evaluation or a smaller distance found
        {
          insert=true;
          k1=kt; // set the smallest distance
          seed1=k;
        }
      }
      // assign i to the cluster ck
      if(seed[s]!=seed1) // if found a nearer seed set it
      {
        seed[s]=seed1;
        change=true; // something has changed
      }
      size[seed[s]]+=1;// increase the size of the cluster

    }
    for(k=0; k<K; k++) // for each cluster recalculate the new mean
    {
      if(size[k]>0)
      {
        for(d=0; d<D; d++)
        {
          M[k*D+d]=0; // set the position to 0 to calculate the new one
          for (s=0; s<S; s++)
          {
            if (seed[s]==k)M[k*D+d]+=P[s*D+d];// for each particle in the cluster add the PB position
          }
          M[k*D+d]=M[k*D+d]/size[k]; // final new position
        }
      }
    }
  }while(change==true && count<=3);

  for(k=0;k<K;k++)
  {
    sigma[k]=0;
    if(size)
    for(s=0; s<S; s++)
    {
      if (seed[s]==k)
      {
        for (d=0; d<D; d++)
        {
          v[d] = P[s*D+d]-M[k*D+d];
        }
        sigma[k]+=inner_product(v, v+D, v, 0.0L); // distance (p-m)^2
      }
    }
    sigma[k]=sigma[k]/(size[k]-1);
  }
  hipMemcpy(d_sigma, sigma, K*sizeof(double), hipMemcpyHostToDevice);
  for(s=0; s<S; s++)
  {
    if(best[seed[s]]==false||fp[s]>fm[seed[s]])
    {
      fm[seed[s]]=fp[s];
      for(d=0; d<D; d++) bp[seed[s]*D+d]=P[s*D+d];
      best[seed[s]]=true;
    }

  }
  hipMemcpy(d_bp, bp, K*D*sizeof(double), hipMemcpyHostToDevice);
}

void update()
{
  int s, d;
  compute<<<NB, TPB>>>(d_V,d_X,d_P,d_seed,d_bp,d_sigma,xmin,xmax,S,D,c,devStates);
  hipMemcpy(X, d_X, S*D*sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(V, d_V, S*D*sizeof(double), hipMemcpyDeviceToHost);

  output1=perf(S,D);
  for (s=0; s<S; s++)
  {
    if(!(output1[s]<= DBL_MAX))output1[s]=0;
    fx[s]=output1[s];
    if (seed[s]!=-1)
    {
      if (fx[s]>fp[s])
      {
        for(d=0; d<D; d++) P[s*D+d]=X[s*D+d];
        fp[s]=fx[s];
        if(fp[s]>fm[seed[s]])
        {
          fm[seed[s]]=fp[s];
          for(d=0; d<D; d++) bp[seed[s]*D+d]=P[s*D+d];
        }
      }

    }
    else
    {
      if (fx[s]>fp[s])
      {
        for(d=0; d<D; d++) P[s*D+d]=X[s*D+d];
        fp[s]=fx[s];

      }
    }
    for (int u=0; u<N; u++)
    {
      if(fx[s]>results[u])
      {
        for(int q=N-1; q>u; q--)
        {
          results[q]=results[q-1];
          g[q]=g[q-1];
        }
        results[u]=fx[s];
        g[u]=group[s];
        break;
      }
      else if(fx[s]==results[u])
      {
        if (g[u]==group[s]) break;
      }
    }


  }
  hipMemcpy(d_P, P, S*D*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_bp, bp, K*D*sizeof(double), hipMemcpyHostToDevice);
}

void identify_niches()
{
  int navg=0; // avarage number of particles per cluster
  int nu;
  double wf; //worst fitness
  double worst=-1; // worst particle
  bool empty;
  int k, s, d;

  for(k=0; k<K; k++)
  {
    navg+=size[k];
  }
  navg=navg/K; // calculate average number of particles per cluster
  nu=0;
  for(k=0; k<K; k++)
  {
    if (size[k]>navg)
    {
      for(int z=0; z<size[k]-navg; z++)
      {
        empty=true;
        wf=0;
        for(s=0;s<S;s++)
        {
          if(seed[s]==k)
          {
            if (fx[s]<wf || empty)
            {
              wf=fx[s];
              worst=s;
              empty=false;
            }
          }
        }
        for(s=worst;s<S;s++) // remove the nj-navg worst particles from cj
        {
          for(d=0; d<D; d++)  X[s*D+d]=X[(s+1)*D+d];

          for(d=0; d<D; d++)  P[s*D+d]=P[(s+1)*D+d];


          for(d=0; d<D; d++)  V[s*D+d]=V[(s+1)*D+d];

          fx[s]=fx[s+1];
          fp[s]=fp[s+1];
          seed[s]=seed[s+1];
          group[s]=group[s+1];
        }
      }
      nu+=size[k]-navg;
      size[k]-=size[k]-navg;
    }
  }
  for(s=S-nu;s<S;s++) // reinitialize the nu un-niched particles
  {
    b=rand()%3;
    if (b==0)
    {

      do
      {
        r1=rand()%D;
        r2=rand()%D;
      }while(r1==r2);
      for ( d = 0; d < D; d++ )
      {

        if(r1==d || r2==d)
        {
          X[s*D+d] = alea( 0, xmax );
        }
        else X[s*D+d] = alea(xmin,0);
        V[s*D+d] = (alea( xmin, xmax ) - X[s*D+d])/2; // Non uniform
      }
    }
    else if(b==1)
    {
      r1=rand()%D;
      for ( d = 0; d < D; d++ )
      {
        X[s*D+d] = alea( xmin, 0);
      }
      for(d=0; d<r1; d++)
      {
        r2=rand()%D;
        X[s*D+r2] = alea(0,xmax);
      }

      for ( d = 0; d < D; d++ )
      {
        V[s*D+d] = (alea( xmin, xmax ) - X[s*D+d])/2; // Non uniform
        P[s*D+d]=X[s*D+d];
      }
    }
    else
    {
      for ( d = 0; d < D; d++ )
      {
        X[s*D+d] = alea( xmin, xmax );
        V[s*D+d] = (alea( xmin, xmax ) - X[s*D+d])/2; // Non uniform
      }

    }

    seed[s]=-1;
  }
  hipMemcpy(d_seed, seed, S*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_X, X, S*D*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_V, V, S*D*sizeof(double), hipMemcpyHostToDevice);
  output1=perf(S,D);
  for (s=0; s<S; s++)
  {
    if(!(output1[s]<= DBL_MAX))output1[s]=0;
    if (seed[s]==-1)
    {
      fx[s]=output1[s];
      for(d=0; d<D; d++) P[s*D+d]=X[s*D+d]; // Best position = current one
      fp[s]=fx[s];
    }

    for (int u=0; u<N; u++)
    {
      if(fx[s]>results[u])
      {
        for(int q=N-1; q>u; q--)
        {
          results[q]=results[q-1];
          g[q]=g[q-1];
        }
        results[u]=fx[s];
        g[u]=group[s];
        break;
      }
      else if(fx[s]==results[u])
      {
        if (g[u]==group[s]) break;
      }
    }

  }
  hipMemcpy(d_P, P, S*D*sizeof(double), hipMemcpyHostToDevice);
}

int main(int argc, const char * argv[]) {
  clock_t tStart = clock();
  int d; // Current dimension
  int s; // Rank of the current particle
  int c1; // intervals for identify niches
  int interv; // print interval
// SC aggiunti per parsing stringa variabili
  char **tokens;
  char *sptr;
  int i, *vars;

  pi = acos( -1 ); // for rastrigin function

 if (argc < 15)
  // SC if (argc < 10)      Mandatory parameters are 14 + the program name
  { // We expect 5 arguments: the program name, the source path and the destination path
    cerr << "Usage: dimension swarm_size  n_seeds  range  n_iterations kmeans_interv print_interv  N_results seed inputfile outputfile zi/tc var_string comp_on hsfile [h_seed]" << endl;
    // SC    cerr << "Usage: dimension swarm_size  n_seeds  range  n_iterations kmeans_interv print_interv  N_results inputfile outputfile" << endl;
    return 1;
  }
  else
  {

    D =atoi(argv[1]); // Search space dimension

// SC aggiunta allocazione vettore di stringhe (nomi variabili)

    if(strlen(argv[13])>0)
    {
	tokens = (char**) calloc(D,sizeof(char*));
	for (i=0; i<D; i++){
	  tokens[i] = (char *) malloc((strlen(argv[13])+2)*sizeof(char));
	  }
    }

vars=(int *) malloc(D*sizeof(int));
sptr = (char *) malloc (strlen(argv[13])*sizeof(char));
strcpy(sptr, argv[13]);

//SC aggiunto parsing stringa variabili
    tokens[0]=strtok((char *) argv[13]," ");
//debug    cout << tokens[0] << " ";
    for (i=1; i<D; i++){
    tokens[i]=strtok(NULL," ");
//debug    cout << tokens[i] << " ";
    }
//debug    cout << "\n\n";


    S=atoi(argv[2]);
    K=atoi(argv[3]);
    x=atof(argv[4]);
    T=atoi(argv[5]);
    c1=atoi(argv[6]);
    interv=atoi(argv[7]);
    N=atoi(argv[8]);
    rseed = (unsigned int) atoi(argv[9]);
    if (argc == 17) {hseed = (int) atoi(argv[16]);}
//  SC    if (argc == 14) {hseed = (int) atoi(argv[13]);}
    else {hseed = (int)rseed;}
  }

  results.resize(N);
  g.resize(N);
  X= (double*) malloc(S*D*sizeof(double));
  V= (double*) malloc(S*D*sizeof(double));
  P= (double*) malloc(S*D*sizeof(double));
  v= (double*) malloc(D*sizeof(double));
  seed=(int*) malloc(S*sizeof(int));
  fx= (double*) malloc(S*sizeof(double));
  fp= (double*) malloc(S*sizeof(double));
  size=(int*) malloc(K*sizeof(int));
  M= (double*) malloc(K*D*sizeof(double));
  bp= (double*) malloc(K*D*sizeof(double));
  sigma= (double*) malloc(K*sizeof(double));
  fm= (double*) malloc(K*sizeof(double));
  best=(bool*) malloc(K*sizeof(bool));

  hipMalloc((void **)&d_X, sizeof(double*)*S*D);
  hipMalloc((void **)&d_V, sizeof(double*)*S*D);
  hipMalloc((void **)&d_P, sizeof(double*)*S*D);
  hipMalloc((void **)&d_seed, sizeof(int*)*S);
  hipMalloc((void **)&d_sigma, sizeof(double*)*K);
  hipMalloc((void **)&d_bp, sizeof(double*)*K*D);
  a=1024/D;
  TPB=512;
  NB=S*D/512;
  hipMalloc ( &devStates, S*D*sizeof( hiprandState ) );

  // ********************************************************************* //
  // INITIALIZATION SECTION - call only once, store app object globally    //
  // ********************************************************************* //

  // create default configuration
  dci::RunInfo configuration = dci::RunInfo();

  // set configuration parameters
  configuration.input_file_name = argv[10];
  string output_file = argv[11];
  configuration.rand_seed = hseed;
  string chosen_index = argv[12];
  if (chosen_index.compare("tc") == 0)
  configuration.tc_index = true;
  else if (chosen_index.compare("zi") == 0)
  configuration.zi_index = true;
  //configuration.hs_input_file_name = "";
  if (chosen_index.compare("tc") == 0)
  configuration.hs_input_file_name = argv[15];

  // create application object
  app = new dci::Application(configuration);

  // initialize application
  app->Init();

  fitness=0;
  w = 0.73;
  c = 2.05;
  // D-cube data
  xmin = -x; xmax = x;

  //-----------------------INITIALIZATION
  setup_kernel <<< NB,TPB >>> ( devStates, (unsigned long) rseed );
  srand(rseed);

  for ( s = 0; s < S; s++ ) // create S particles
  {
    b = rand()%3;

    if (b==0)
    {
      do
      {
        r1 = rand()%D;
        r2 = rand()%D;
      }while(r1==r2);
      for (d = 0; d < D; d++)
      {
        if(r1==d || r2==d)
        {
          X[s*D+d] = alea( 0, xmax );
        }
        else
        X[s*D+d] = alea(xmin,0);
        V[s*D+d] = (alea( xmin, xmax ) - X[s*D+d])/2; // Non uniform
        P[s*D+d] = X[s*D+d];
      }
    }
    else if(b==1)
    {
      r1 = rand()%D;
      for (d = 0; d < D; d++)
      {
        X[s*D+d] = alea(xmin, 0);
      }
      for(d=0; d<r1; d++)
      {
        r2 = rand()%D;
        X[s*D+r2] = alea(0,xmax);
      }

      for (d = 0; d < D; d++)
      {
        V[s*D+d] = (alea( xmin, xmax ) - X[s*D+d])/2; // Non uniform
        P[s*D+d] = X[s*D+d];
      }
    }
    else
    {
      for (d = 0; d < D; d++)
      {
        X[s*D+d] = alea( xmin, xmax );
        V[s*D+d] = (alea( xmin, xmax ) - X[s*D+d])/2; // Non uniform
        P[s*D+d] = X[s*D+d];
      }
    }
  }
  hipMemcpy(d_X, X, S*D*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_V, V, S*D*sizeof(double), hipMemcpyHostToDevice);
  output1 = perf(S,D);
  for (s=0; s<S; s++)
  {
    if(!(output1[s] <= DBL_MAX))output1[s]=0;
    fx[s] = output1[s];
    fp[s] = fx[s];
    for (int u=0; u<N; u++)
    {
      if(fx[s]>results[u])
      {
        for(int q=N-1; q>u; q--)
        {
          results[q]=results[q-1];
          g[q]=g[q-1];
        }
        results[u]=fx[s];
        g[u]=group[s];
        break;
      }
      else if(fx[s]==results[u])
      {
        if (g[u]==group[s]) break;
      }
    }
  }
  hipMemcpy(d_P, P, S*D*sizeof(double), hipMemcpyHostToDevice);

  k_means(); //k-means algorithm
  hipMemcpy(d_seed, seed, S*sizeof(int), hipMemcpyHostToDevice);

  //--------------------ITERATIONS
  for (int t=1; t<T; t++)
  {
    update();
    if (t % c1 ==0)
    {
      k_means();
      identify_niches();
    }


// SC Separato l'output su video (???) e quello su file
    //PRINT ON SCREEN
    std::ofstream outfile2;


    int var_count = 0;
    if(t%interv==0 || t==T-1){
//  SC Aggiunta per stampare l'intestazione del file risultati

if(strlen(sptr)>1)
{
 for(i=0;i<D;i++)
  outfile2 << tokens[i] << "\t";

if(atoi(argv[14])==0)
 outfile2 << argv[12] << "\n";
    else
 outfile2 << argv[12] << "\tComp\n";
}

    for(int u=0; u<N;u++) {
        for(i=0;i<D;i++) vars[i]=-1;
        int vcount=0;
        for (d=0; d<g[u].size(); d++) {
          for (i=var_count; i<g[u][d]; i++){
            outfile2 << "0" << "\t";
//debug    cout << "0" << "\t";
	    }
          outfile2 << "1" << "\t";
//debug	  cout << "1" << "\t";
	  vars[vcount]=i;
          var_count = g[u][d]+1;
	  vcount=vcount+1;
        }
        while (var_count < D) {
          outfile2 << "0" << "\t";
//debug	  cout << "0" << "\t";
          var_count++;
        }

//debug        i=0;
//debug	while (vars[i]>=0) {cout << tokens[vars[i]] << " ";i++;}
//debug        cout << "\n";

// SC	outfile << results[u]<< "\n";
// SC   outfile2 << results[u]<< "\n";

       if(atoi(argv[14])==0)
       {
	outfile2 << results[u]<< "\n";
       }
       else
       {
	outfile2 << results[u]<< "\t";
        int  nv=0;
       while (vars[nv]>=0) nv++;
       for(i=0;i<nv-1;i++) outfile2 << tokens[vars[i]] << "+";
       outfile2 << tokens[vars[nv-1]] << "\n";
       }
        var_count = 0;
      }
      cout << "fitness computed " << fitness << " times\n";
      cout << "Time taken: " << (double)(clock() - tStart)/CLOCKS_PER_SEC << "s\n";
      cout <<"------------------------\n\n";

    }

   //PRINT ON FILE

  if(t==T-1)
  {
    std::ofstream outfile;

    outfile.open(output_file, std::ios_base::app);

    var_count = 0;

//  SC Aggiunta per stampare l'intestazione del file risultati

if(strlen(sptr)>1)
{
 for(i=0;i<D;i++){
  outfile << tokens[i] << "\t";
//debug cout << tokens[i] << "\t";
  }

if(atoi(argv[14])==0)
 {outfile << argv[12] << "\n";
//debug  cout  << argv[12] << "\n";
  }
    else
 {outfile << argv[12] << "\tComp\n";
//debug cout <<  argv[12] << "\tComp\n";
}

}
      for(int u=0; u<N;u++) {
        for(i=0;i<D;i++) vars[i]=-1;
        int vcount=0;
        for (d=0; d<g[u].size(); d++) {
          for (i=var_count; i<g[u][d]; i++){
            outfile << "0" << "\t";
//debug    cout << "0" << "\t";
	    }
          outfile << "1" << "\t";
//debug	  cout << "1" << "\t";
	  vars[vcount]=i;
          var_count = g[u][d]+1;
	  vcount=vcount+1;
        }
        while (var_count < D) {
          outfile << "0" << "\t";
//debug	  cout << "0" << "\t";
          var_count++;
        }

//debug        i=0;
//debug	while (vars[i]>=0) {cout << tokens[vars[i]] << " ";i++;}
//debug        cout << "\n";

// SC	outfile << results[u]<< "\n";
// SC   outfile2 << results[u]<< "\n";

       if(atoi(argv[14])==0)
       {
	outfile << results[u]<< "\n";
       }
// SC Stampa le variabili composte
       else
       {
	outfile << results[u]<< "\t";
        int  nv=0;
       while (vars[nv]>=0) nv++;
       for(i=0;i<nv-1;i++) outfile << tokens[vars[i]] << "+";
       outfile << tokens[vars[nv-1]] << "\n";
       }
        var_count = 0;
      }
    outfile.close();
    }


   outfile2.close();

  }
  // delete app object
  hipFree(d_X);
  hipFree(d_V);
  hipFree(d_P);
  hipFree(d_bp);
  hipFree(d_seed);
  hipFree(d_sigma);
  delete app;

  return 0;
}

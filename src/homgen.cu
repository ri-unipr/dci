#include "hip/hip_runtime.h"

/*
* File:   hom_gen.cu
* Author: Michele Amoretti
*/

#include <cstdlib>
#include "application.h"

using namespace std;

/*
* Entry point: retrieves configuration from command-line parameters
* and runs main application if a good configuration was supplied.
*/
int main(int argc, char** argv) {

  // process command line to obtain configuration
  //dci::RunInfo_p configuration = dci::processCommandLine(argc, argv);
  dci::RunInfo_p configuration(new dci::RunInfo());
  int pos;

  // cycle all command-line arguments
  for (int i = 1; i < argc; ++i)
  {
    std::string arg(argv[i]);

    /*
    * input filename retrieval
    */
    if (arg.length() >= 2 && (arg[0] != '-' || arg[1] != '-')) // this is a filename
    {
      if (configuration->input_file_name != "") // file was already specified, error
      {
        configuration->error_message = "more than one file specified or unknown argument: " + arg;
        cout << "Error: " << configuration->error_message << "\n\n";
        return -1;
      }
      configuration->input_file_name = arg;
    }
    else if (arg.length() >= 2 && (pos = arg.find_first_of(':')) != std::string::npos) // this is a --XXX:VVV parameter
    {

      if (pos == arg.length() - 1) // empty argument value
      {
        configuration->error_message = "no value specified for argument: " + arg;
        cout << "Error: " << configuration->error_message << "\n\n";
        return -1;
      }

      std::string name = arg.substr(0, pos);
      std::string value = arg.substr(pos + 1);

      if (name == "--hs-out")
      configuration->hs_output_file_name = value;
      else if (name == "--hs-data-out")
      configuration->hs_data_output_file_name = value;
      /*
      else if (name == "--rand-seed")
      {
        if ((configuration->rand_seed = std::atoi(value.data())) == 0)
        {
          configuration->error_message = "rand seed invalid or zero: " + value;
          return -1;
        }
      }
      */
      else
      {
        configuration->error_message = "unknown argument: " + name;
        cout << "Error: " << configuration->error_message << "\n\n";
        return -1;
      }
    }
    /*
    * unknown parameters
    */
    else
    {
      configuration->error_message = "unknown argument: " + arg;
      cout << "Error: " << configuration->error_message << "\n\n";
      return -1;
    }
  }

  configuration->verbose = true;
  configuration->tc_index = true;
  srand(time(NULL));
  configuration->rand_seed = rand();

  // store start/end time
  clock_t start = clock(), stop;

  // create application object
  dci::Application* app = new dci::Application(*configuration);

  app->init(); // initialize application

  // get end time
  stop = clock();

  // get duration
  cout << "Computing time:  " << app->elapsedTimeMilliseconds(start, stop) << " ms" << endl;

  delete app;

  return 1;

}
